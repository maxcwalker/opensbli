// Auto-generated at 2024-09-02 15:36:03.363180 by ops-translator legacy

//header
#define OPS_API 2
#define OPS_3D
#include "ops_lib_core.h"

#include "ops_cuda_rt_support.h"
#include "ops_cuda_reduction.h"

#include <hip/hip_complex.h>

#ifdef OPS_MPI
#include "ops_mpi_core.h"
#endif
// global constants
__constant__ double Delta0block0;
__constant__ double Delta1block0;
__constant__ double Delta2block0;
__constant__ int HDF5_timing;
__constant__ double Minf;
__constant__ double Pr;
__constant__ double Re;
__constant__ double RefT;
__constant__ double SuthT;
__constant__ double Twall;
__constant__ double b_f;
__constant__ double beta_0;
__constant__ int block0np0;
__constant__ int block0np1;
__constant__ int block0np2;
__constant__ double dt;
__constant__ double gama;
__constant__ double gamma_m1;
__constant__ double inv2Delta0block0;
__constant__ double inv2Delta1block0;
__constant__ double inv2Delta2block0;
__constant__ double inv2Minf;
__constant__ double invDelta0block0;
__constant__ double invDelta1block0;
__constant__ double invDelta2block0;
__constant__ double invPr;
__constant__ double invRe;
__constant__ double invRefT;
__constant__ double inv_gamma_m1;
__constant__ double invgama;
__constant__ double invgamma_m1;
__constant__ int niter;
__constant__ double omega_0;
__constant__ double shock_filter_control;
__constant__ double simulation_time;
__constant__ int start_iter;
__constant__ double tripA;
__constant__ int write_output_file;
__constant__ double xts;

void ops_init_backend() {}

void ops_decl_const_char(OPS_instance *instance, int dim, char const *type,
int size, char *dat, char const *name){
  ops_execute(instance);
  if (!strcmp(name,"Delta0block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Delta0block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"Delta1block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Delta1block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"Delta2block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Delta2block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"HDF5_timing")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(HDF5_timing), dat, dim*size));
  }
  else
  if (!strcmp(name,"Minf")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Minf), dat, dim*size));
  }
  else
  if (!strcmp(name,"Pr")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Pr), dat, dim*size));
  }
  else
  if (!strcmp(name,"Re")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Re), dat, dim*size));
  }
  else
  if (!strcmp(name,"RefT")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(RefT), dat, dim*size));
  }
  else
  if (!strcmp(name,"SuthT")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(SuthT), dat, dim*size));
  }
  else
  if (!strcmp(name,"Twall")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Twall), dat, dim*size));
  }
  else
  if (!strcmp(name,"b_f")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(b_f), dat, dim*size));
  }
  else
  if (!strcmp(name,"beta_0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(beta_0), dat, dim*size));
  }
  else
  if (!strcmp(name,"block0np0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(block0np0), dat, dim*size));
  }
  else
  if (!strcmp(name,"block0np1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(block0np1), dat, dim*size));
  }
  else
  if (!strcmp(name,"block0np2")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(block0np2), dat, dim*size));
  }
  else
  if (!strcmp(name,"dt")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(dt), dat, dim*size));
  }
  else
  if (!strcmp(name,"gama")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(gama), dat, dim*size));
  }
  else
  if (!strcmp(name,"gamma_m1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(gamma_m1), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Delta0block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Delta0block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Delta1block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Delta1block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Delta2block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Delta2block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Minf")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Minf), dat, dim*size));
  }
  else
  if (!strcmp(name,"invDelta0block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invDelta0block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"invDelta1block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invDelta1block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"invDelta2block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invDelta2block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"invPr")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invPr), dat, dim*size));
  }
  else
  if (!strcmp(name,"invRe")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invRe), dat, dim*size));
  }
  else
  if (!strcmp(name,"invRefT")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invRefT), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv_gamma_m1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv_gamma_m1), dat, dim*size));
  }
  else
  if (!strcmp(name,"invgama")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invgama), dat, dim*size));
  }
  else
  if (!strcmp(name,"invgamma_m1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invgamma_m1), dat, dim*size));
  }
  else
  if (!strcmp(name,"niter")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(niter), dat, dim*size));
  }
  else
  if (!strcmp(name,"omega_0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(omega_0), dat, dim*size));
  }
  else
  if (!strcmp(name,"shock_filter_control")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(shock_filter_control), dat, dim*size));
  }
  else
  if (!strcmp(name,"simulation_time")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(simulation_time), dat, dim*size));
  }
  else
  if (!strcmp(name,"start_iter")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(start_iter), dat, dim*size));
  }
  else
  if (!strcmp(name,"tripA")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(tripA), dat, dim*size));
  }
  else
  if (!strcmp(name,"write_output_file")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(write_output_file), dat, dim*size));
  }
  else
  if (!strcmp(name,"xts")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(xts), dat, dim*size));
  }
  else
  {
    throw OPSException(OPS_RUNTIME_ERROR, "error: unknown const name");
}
}


//user kernel files
#include "opensbliblock00Kernel067_kernel.cu"
#include "opensbliblock00Kernel070_kernel.cu"
#include "opensbliblock00Kernel071_kernel.cu"
#include "opensbliblock00Kernel072_kernel.cu"
#include "opensbliblock00Kernel073_kernel.cu"
#include "opensbliblock00Kernel074_kernel.cu"
#include "opensbliblock00Kernel081_kernel.cu"
#include "opensbliblock00Kernel061_kernel.cu"
#include "opensbliblock00Kernel062_kernel.cu"
#include "opensbliblock00Kernel063_kernel.cu"
#include "opensbliblock00Kernel064_kernel.cu"
#include "opensbliblock00Kernel003_kernel.cu"
#include "opensbliblock00Kernel007_kernel.cu"
#include "opensbliblock00Kernel012_kernel.cu"
#include "opensbliblock00Kernel004_kernel.cu"
#include "opensbliblock00Kernel009_kernel.cu"
#include "opensbliblock00Kernel018_kernel.cu"
#include "opensbliblock00Kernel020_kernel.cu"
#include "opensbliblock00Kernel011_kernel.cu"
#include "opensbliblock00Kernel030_kernel.cu"
#include "opensbliblock00Kernel022_kernel.cu"
#include "opensbliblock00Kernel000_kernel.cu"
#include "opensbliblock00Kernel001_kernel.cu"
#include "opensbliblock00Kernel002_kernel.cu"
#include "opensbliblock00Kernel021_kernel.cu"
#include "opensbliblock00Kernel032_kernel.cu"
#include "opensbliblock00Kernel034_kernel.cu"
#include "opensbliblock00Kernel036_kernel.cu"
#include "opensbliblock00Kernel038_kernel.cu"
#include "opensbliblock00Kernel040_kernel.cu"
#include "opensbliblock00Kernel041_kernel.cu"
#include "opensbliblock00Kernel042_kernel.cu"
#include "opensbliblock00Kernel043_kernel.cu"
#include "opensbliblock00Kernel044_kernel.cu"
#include "opensbliblock00Kernel045_kernel.cu"
#include "opensbliblock00Kernel046_kernel.cu"
#include "opensbliblock00Kernel047_kernel.cu"
#include "opensbliblock00Kernel056_kernel.cu"
#include "opensbliblock00Kernel082_kernel.cu"
