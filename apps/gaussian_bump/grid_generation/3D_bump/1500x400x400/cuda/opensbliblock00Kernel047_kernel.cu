#include "hip/hip_runtime.h"
// Auto-generated at 2024-09-02 15:36:03.344289 by ops-translator legacy

__constant__ int dims_opensbliblock00Kernel047 [2][2];
static int dims_opensbliblock00Kernel047_h [2][2] = {{0}};

//  =============
//  User function
//  =============
__device__ void opensbliblock00Kernel047_gpu(const ACC<double> &T_B0,
  ACC<double> &wk11_B0)
{
    wk11_B0(0,0,0) = (-(2.0/3.0)*T_B0(0,0,-1) - (1.0/12.0)*T_B0(0,0,2) + ((1.0/12.0))*T_B0(0,0,-2) +
      ((2.0/3.0))*T_B0(0,0,1))*invDelta2block0;

}


//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_opensbliblock00Kernel047(
double* __restrict arg0,
double* __restrict arg1,
int size0,
int size1,
int size2) {
    int idx_z = blockDim.z * blockIdx.z + threadIdx.z;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    arg0 += idx_x * 1*1 + idx_y * 1*1 * dims_opensbliblock00Kernel047[0][0] + idx_z * 1*1 * dims_opensbliblock00Kernel047[0][0] * dims_opensbliblock00Kernel047[0][1];
    arg1 += idx_x * 1*1 + idx_y * 1*1 * dims_opensbliblock00Kernel047[1][0] + idx_z * 1*1 * dims_opensbliblock00Kernel047[1][0] * dims_opensbliblock00Kernel047[1][1];

    if (idx_x < size0 && idx_y < size1 && idx_z < size2) {
        const ACC<double> argp0(dims_opensbliblock00Kernel047[0][0], dims_opensbliblock00Kernel047[0][1], arg0);
        ACC<double> argp1(dims_opensbliblock00Kernel047[1][0], dims_opensbliblock00Kernel047[1][1], arg1);
        opensbliblock00Kernel047_gpu(
         argp0, argp1);
    }

} //End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_opensbliblock00Kernel047(char const *name, ops_block block, int dim, int* range,
 ops_arg arg0, ops_arg arg1) {
#else
void ops_par_loop_opensbliblock00Kernel047_execute(ops_kernel_descriptor *desc) {
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[2];

    args[0] = arg0;
    args[1] = arg1;

#if CHECKPOINTING && !OPS_LAZY
    if (!ops_checkpointing_before(args, 2, range, 36)) return;
#endif

    if (block->instance->OPS_diags > 1) {
        ops_timing_realloc(block->instance, 36, "opensbliblock00Kernel047");
        block->instance->OPS_kernels[36].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[3];
    int end_indx[3];
#ifdef OPS_MPI
    int arg_idx[3];
#endif

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 3; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 2, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

    int xdim0 = args[0].dat->size[0];
    int ydim0 = args[0].dat->size[1];
    int xdim1 = args[1].dat->size[0];
    int ydim1 = args[1].dat->size[1];

    if (xdim0 != dims_opensbliblock00Kernel047_h[0][0] || ydim0 != dims_opensbliblock00Kernel047_h[0][1] || xdim1 != dims_opensbliblock00Kernel047_h[1][0] || ydim1 != dims_opensbliblock00Kernel047_h[1][1]) {
        dims_opensbliblock00Kernel047_h[0][0] = xdim0;
        dims_opensbliblock00Kernel047_h[0][1] = ydim0;
        dims_opensbliblock00Kernel047_h[1][0] = xdim1;
        dims_opensbliblock00Kernel047_h[1][1] = ydim1;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_opensbliblock00Kernel047), dims_opensbliblock00Kernel047_h, sizeof(dims_opensbliblock00Kernel047)));
    }


    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);
    int z_size = MAX(0,end_indx[2]-start_indx[2]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x+ 1, (y_size-1)/block->instance->OPS_block_size_y + 1, (z_size-1)/block->instance->OPS_block_size_z +1);
    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);



    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);
    long long int dat1 = (block->instance->OPS_soa ? args[1].dat->type_size : args[1].dat->elem_size);

    char *p_a[2];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + 
             dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 *
                       args[0].dat->size[0] *
                      (start_indx[1] * args[0].stencil->stride[1]);
    base0 = base0 + dat0 *
                       args[0].dat->size[0] *
                       args[0].dat->size[1] *
                      (start_indx[2] * args[0].stencil->stride[2]);
    p_a[0] = (char *)args[0].data_d + base0;

    long long int base1 = args[1].dat->base_offset + 
             dat1 * 1 * (start_indx[0] * args[1].stencil->stride[0]);
    base1 = base1 + dat1 *
                       args[1].dat->size[0] *
                      (start_indx[1] * args[1].stencil->stride[1]);
    base1 = base1 + dat1 *
                       args[1].dat->size[0] *
                       args[1].dat->size[1] *
                      (start_indx[2] * args[1].stencil->stride[2]);
    p_a[1] = (char *)args[1].data_d + base1;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 2);
    ops_halo_exchanges(args, 2, range);
#endif

    if (block->instance->OPS_diags > 1) {
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[36].mpi_time += __t2 - __t1;
    }


//call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0 && z_size > 0) {

        ops_opensbliblock00Kernel047<<<grid, tblock >>> ( 
         (double *)p_a[0], (double *)p_a[1],
        x_size, y_size, z_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if (block->instance->OPS_diags>1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[36].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 2);
    ops_set_halo_dirtybit3(&args[1], range);
#endif

    if (block->instance->OPS_diags > 1) {
        //  ====================
        //  Update kernel record
        //  ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[36].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[36].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
        block->instance->OPS_kernels[36].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg1);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_opensbliblock00Kernel047(char const *name, ops_block block, int dim, int* range,
 ops_arg arg0, ops_arg arg1)
{
    ops_arg args[2];

    args[0] = arg0;
    args[1] = arg1;

    create_kerneldesc_and_enque("opensbliblock00Kernel047", args, 2, 36, dim, 1, range, block, ops_par_loop_opensbliblock00Kernel047_execute);
}
#endif
