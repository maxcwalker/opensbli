// Auto-generated at 2024-10-10 09:28:37.151234 by ops-translator legacy

//header
#define OPS_API 2
#define OPS_3D
#include "ops_lib_core.h"

#include "ops_cuda_rt_support.h"
#include "ops_cuda_reduction.h"

#include <hip/hip_complex.h>

#ifdef OPS_MPI
#include "ops_mpi_core.h"
#endif
// global constants
__constant__ double Delta0block0;
__constant__ double Delta1block0;
__constant__ double Delta2block0;
__constant__ double H;
__constant__ int HDF5_timing;
__constant__ double L;
__constant__ double Minf;
__constant__ double Pr;
__constant__ double Re;
__constant__ double RefT;
__constant__ double SuthT;
__constant__ double Twall;
__constant__ double a;
__constant__ double b;
__constant__ double b_f;
__constant__ double beta_0;
__constant__ int block0np0;
__constant__ int block0np1;
__constant__ int block0np2;
__constant__ double dt;
__constant__ double gama;
__constant__ double gamma_m1;
__constant__ double inv2Delta0block0;
__constant__ double inv2Delta1block0;
__constant__ double inv2Delta2block0;
__constant__ double inv2Minf;
__constant__ double inv2a;
__constant__ double invDelta0block0;
__constant__ double invDelta1block0;
__constant__ double invDelta2block0;
__constant__ double invH;
__constant__ double invPr;
__constant__ double invRe;
__constant__ double invRefT;
__constant__ double inv_gamma_m1;
__constant__ double invgama;
__constant__ double invgamma_m1;
__constant__ double invnsamples;
__constant__ int niter;
__constant__ int nsamples;
__constant__ double omega_0;
__constant__ double omega_1;
__constant__ double omega_2;
__constant__ double omega_3;
__constant__ double phi_0;
__constant__ double phi_1;
__constant__ double phi_2;
__constant__ double shock_filter_control;
__constant__ double simulation_time;
__constant__ int start_iter;
__constant__ int stat_frequency;
__constant__ double tripA;
__constant__ int write_output_file;
__constant__ int write_slices;
__constant__ double xts;

void ops_init_backend() {}

void ops_decl_const_char(OPS_instance *instance, int dim, char const *type,
int size, char *dat, char const *name){
  ops_execute(instance);
  if (!strcmp(name,"Delta0block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Delta0block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"Delta1block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Delta1block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"Delta2block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Delta2block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"H")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(H), dat, dim*size));
  }
  else
  if (!strcmp(name,"HDF5_timing")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(HDF5_timing), dat, dim*size));
  }
  else
  if (!strcmp(name,"L")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(L), dat, dim*size));
  }
  else
  if (!strcmp(name,"Minf")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Minf), dat, dim*size));
  }
  else
  if (!strcmp(name,"Pr")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Pr), dat, dim*size));
  }
  else
  if (!strcmp(name,"Re")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Re), dat, dim*size));
  }
  else
  if (!strcmp(name,"RefT")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(RefT), dat, dim*size));
  }
  else
  if (!strcmp(name,"SuthT")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(SuthT), dat, dim*size));
  }
  else
  if (!strcmp(name,"Twall")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(Twall), dat, dim*size));
  }
  else
  if (!strcmp(name,"a")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(a), dat, dim*size));
  }
  else
  if (!strcmp(name,"b")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(b), dat, dim*size));
  }
  else
  if (!strcmp(name,"b_f")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(b_f), dat, dim*size));
  }
  else
  if (!strcmp(name,"beta_0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(beta_0), dat, dim*size));
  }
  else
  if (!strcmp(name,"block0np0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(block0np0), dat, dim*size));
  }
  else
  if (!strcmp(name,"block0np1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(block0np1), dat, dim*size));
  }
  else
  if (!strcmp(name,"block0np2")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(block0np2), dat, dim*size));
  }
  else
  if (!strcmp(name,"dt")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(dt), dat, dim*size));
  }
  else
  if (!strcmp(name,"gama")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(gama), dat, dim*size));
  }
  else
  if (!strcmp(name,"gamma_m1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(gamma_m1), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Delta0block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Delta0block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Delta1block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Delta1block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Delta2block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Delta2block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2Minf")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2Minf), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv2a")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv2a), dat, dim*size));
  }
  else
  if (!strcmp(name,"invDelta0block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invDelta0block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"invDelta1block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invDelta1block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"invDelta2block0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invDelta2block0), dat, dim*size));
  }
  else
  if (!strcmp(name,"invH")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invH), dat, dim*size));
  }
  else
  if (!strcmp(name,"invPr")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invPr), dat, dim*size));
  }
  else
  if (!strcmp(name,"invRe")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invRe), dat, dim*size));
  }
  else
  if (!strcmp(name,"invRefT")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invRefT), dat, dim*size));
  }
  else
  if (!strcmp(name,"inv_gamma_m1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(inv_gamma_m1), dat, dim*size));
  }
  else
  if (!strcmp(name,"invgama")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invgama), dat, dim*size));
  }
  else
  if (!strcmp(name,"invgamma_m1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invgamma_m1), dat, dim*size));
  }
  else
  if (!strcmp(name,"invnsamples")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(invnsamples), dat, dim*size));
  }
  else
  if (!strcmp(name,"niter")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(niter), dat, dim*size));
  }
  else
  if (!strcmp(name,"nsamples")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(nsamples), dat, dim*size));
  }
  else
  if (!strcmp(name,"omega_0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(omega_0), dat, dim*size));
  }
  else
  if (!strcmp(name,"omega_1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(omega_1), dat, dim*size));
  }
  else
  if (!strcmp(name,"omega_2")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(omega_2), dat, dim*size));
  }
  else
  if (!strcmp(name,"omega_3")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(omega_3), dat, dim*size));
  }
  else
  if (!strcmp(name,"phi_0")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(phi_0), dat, dim*size));
  }
  else
  if (!strcmp(name,"phi_1")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(phi_1), dat, dim*size));
  }
  else
  if (!strcmp(name,"phi_2")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(phi_2), dat, dim*size));
  }
  else
  if (!strcmp(name,"shock_filter_control")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(shock_filter_control), dat, dim*size));
  }
  else
  if (!strcmp(name,"simulation_time")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(simulation_time), dat, dim*size));
  }
  else
  if (!strcmp(name,"start_iter")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(start_iter), dat, dim*size));
  }
  else
  if (!strcmp(name,"stat_frequency")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(stat_frequency), dat, dim*size));
  }
  else
  if (!strcmp(name,"tripA")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(tripA), dat, dim*size));
  }
  else
  if (!strcmp(name,"write_output_file")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(write_output_file), dat, dim*size));
  }
  else
  if (!strcmp(name,"write_slices")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(write_slices), dat, dim*size));
  }
  else
  if (!strcmp(name,"xts")) {
      cutilSafeCall(instance->ostream(),hipMemcpyToSymbol(HIP_SYMBOL(xts), dat, dim*size));
  }
  else
  {
    throw OPSException(OPS_RUNTIME_ERROR, "error: unknown const name");
}
}


//user kernel files
#include "opensbliblock00Kernel067_kernel.cu"
#include "opensbliblock00Kernel070_kernel.cu"
#include "opensbliblock00Kernel071_kernel.cu"
#include "opensbliblock00Kernel072_kernel.cu"
#include "opensbliblock00Kernel073_kernel.cu"
#include "opensbliblock00Kernel074_kernel.cu"
#include "opensbliblock00Kernel081_kernel.cu"
#include "opensbliblock00Kernel061_kernel.cu"
#include "opensbliblock00Kernel062_kernel.cu"
#include "opensbliblock00Kernel063_kernel.cu"
#include "opensbliblock00Kernel064_kernel.cu"
#include "opensbliblock00Kernel006_kernel.cu"
#include "opensbliblock00Kernel008_kernel.cu"
#include "opensbliblock00Kernel016_kernel.cu"
#include "opensbliblock00Kernel011_kernel.cu"
#include "opensbliblock00Kernel012_kernel.cu"
#include "opensbliblock00Kernel017_kernel.cu"
#include "opensbliblock00Kernel020_kernel.cu"
#include "opensbliblock00Kernel010_kernel.cu"
#include "opensbliblock00Kernel027_kernel.cu"
#include "opensbliblock00Kernel022_kernel.cu"
#include "opensbliblock00Kernel000_kernel.cu"
#include "opensbliblock00Kernel001_kernel.cu"
#include "opensbliblock00Kernel002_kernel.cu"
#include "opensbliblock00Kernel021_kernel.cu"
#include "opensbliblock00Kernel032_kernel.cu"
#include "opensbliblock00Kernel034_kernel.cu"
#include "opensbliblock00Kernel036_kernel.cu"
#include "opensbliblock00Kernel038_kernel.cu"
#include "opensbliblock00Kernel040_kernel.cu"
#include "opensbliblock00Kernel041_kernel.cu"
#include "opensbliblock00Kernel042_kernel.cu"
#include "opensbliblock00Kernel043_kernel.cu"
#include "opensbliblock00Kernel044_kernel.cu"
#include "opensbliblock00Kernel045_kernel.cu"
#include "opensbliblock00Kernel046_kernel.cu"
#include "opensbliblock00Kernel047_kernel.cu"
#include "opensbliblock00Kernel056_kernel.cu"
#include "opensbliblock00Kernel084_kernel.cu"
#include "opensbliblock00Kernel082_kernel.cu"
#include "monitor_0_p_B0_kernel.cu"
#include "monitor_1_p_B0_kernel.cu"
#include "monitor_2_p_B0_kernel.cu"
#include "monitor_3_p_B0_kernel.cu"
#include "monitor_4_p_B0_kernel.cu"
#include "monitor_5_p_B0_kernel.cu"
#include "monitor_6_p_B0_kernel.cu"
#include "monitor_7_p_B0_kernel.cu"
#include "monitor_8_p_B0_kernel.cu"
#include "monitor_9_p_B0_kernel.cu"
#include "monitor_10_p_B0_kernel.cu"
#include "monitor_11_p_B0_kernel.cu"
#include "monitor_12_p_B0_kernel.cu"
#include "monitor_13_p_B0_kernel.cu"
#include "monitor_14_p_B0_kernel.cu"
#include "monitor_15_p_B0_kernel.cu"
#include "monitor_16_p_B0_kernel.cu"
#include "monitor_17_p_B0_kernel.cu"
#include "monitor_18_p_B0_kernel.cu"
#include "monitor_19_p_B0_kernel.cu"
#include "monitor_20_p_B0_kernel.cu"
#include "monitor_21_p_B0_kernel.cu"
#include "monitor_22_p_B0_kernel.cu"
#include "monitor_23_p_B0_kernel.cu"
#include "monitor_24_p_B0_kernel.cu"
#include "monitor_25_p_B0_kernel.cu"
#include "monitor_26_p_B0_kernel.cu"
#include "monitor_27_p_B0_kernel.cu"
#include "monitor_28_p_B0_kernel.cu"
#include "monitor_29_p_B0_kernel.cu"
#include "monitor_30_p_B0_kernel.cu"
#include "monitor_31_p_B0_kernel.cu"
#include "monitor_32_p_B0_kernel.cu"
#include "monitor_33_p_B0_kernel.cu"
#include "monitor_34_p_B0_kernel.cu"
#include "monitor_35_p_B0_kernel.cu"
#include "monitor_36_p_B0_kernel.cu"
#include "monitor_37_rhou0_B0_kernel.cu"
#include "monitor_38_rhou0_B0_kernel.cu"
#include "monitor_39_rhou0_B0_kernel.cu"
#include "monitor_40_rhou0_B0_kernel.cu"
#include "monitor_41_rhou0_B0_kernel.cu"
#include "monitor_42_rhou0_B0_kernel.cu"
#include "monitor_43_rhou0_B0_kernel.cu"
#include "monitor_44_rhou0_B0_kernel.cu"
#include "monitor_45_rhou0_B0_kernel.cu"
#include "monitor_46_rhou0_B0_kernel.cu"
#include "monitor_47_rhou0_B0_kernel.cu"
#include "monitor_48_rhou0_B0_kernel.cu"
#include "monitor_49_rhou0_B0_kernel.cu"
#include "monitor_50_rhou0_B0_kernel.cu"
#include "monitor_51_rhou0_B0_kernel.cu"
#include "monitor_52_rhou0_B0_kernel.cu"
#include "monitor_53_rhou0_B0_kernel.cu"
#include "monitor_54_rhou0_B0_kernel.cu"
#include "monitor_55_rhou0_B0_kernel.cu"
#include "monitor_56_rhou0_B0_kernel.cu"
#include "monitor_57_rhou0_B0_kernel.cu"
#include "monitor_58_rhou0_B0_kernel.cu"
#include "monitor_59_rhou0_B0_kernel.cu"
#include "monitor_60_rhou0_B0_kernel.cu"
#include "monitor_61_rhou0_B0_kernel.cu"
#include "monitor_62_rhou0_B0_kernel.cu"
#include "monitor_63_rhou0_B0_kernel.cu"
#include "monitor_64_rhou0_B0_kernel.cu"
#include "monitor_65_rhou0_B0_kernel.cu"
#include "monitor_66_rhou0_B0_kernel.cu"
#include "monitor_67_rhou0_B0_kernel.cu"
#include "monitor_68_rhou0_B0_kernel.cu"
#include "monitor_69_rhou0_B0_kernel.cu"
#include "monitor_70_rhou0_B0_kernel.cu"
#include "monitor_71_rhou0_B0_kernel.cu"
#include "monitor_72_rhou0_B0_kernel.cu"
#include "monitor_73_rhou0_B0_kernel.cu"
#include "monitor_74_rhou2_B0_kernel.cu"
#include "monitor_75_rhou2_B0_kernel.cu"
#include "monitor_76_rhou2_B0_kernel.cu"
#include "monitor_77_rhou2_B0_kernel.cu"
#include "monitor_78_rhou2_B0_kernel.cu"
#include "monitor_79_rhou2_B0_kernel.cu"
#include "monitor_80_rhou2_B0_kernel.cu"
#include "monitor_81_rhou2_B0_kernel.cu"
#include "monitor_82_rhou2_B0_kernel.cu"
#include "monitor_83_rhou2_B0_kernel.cu"
#include "monitor_84_rhou2_B0_kernel.cu"
#include "monitor_85_rhou2_B0_kernel.cu"
#include "monitor_86_rhou2_B0_kernel.cu"
#include "monitor_87_rhou2_B0_kernel.cu"
#include "monitor_88_rhou2_B0_kernel.cu"
#include "monitor_89_rhou2_B0_kernel.cu"
#include "monitor_90_rhou2_B0_kernel.cu"
#include "monitor_91_rhou2_B0_kernel.cu"
#include "monitor_92_rhou2_B0_kernel.cu"
#include "monitor_93_rhou2_B0_kernel.cu"
#include "monitor_94_rhou2_B0_kernel.cu"
#include "monitor_95_rhou2_B0_kernel.cu"
#include "monitor_96_rhou2_B0_kernel.cu"
#include "monitor_97_rhou2_B0_kernel.cu"
#include "monitor_98_rhou2_B0_kernel.cu"
#include "monitor_99_rhou2_B0_kernel.cu"
#include "monitor_100_rhou2_B0_kernel.cu"
#include "monitor_101_rhou2_B0_kernel.cu"
#include "monitor_102_rhou2_B0_kernel.cu"
#include "monitor_103_rhou2_B0_kernel.cu"
#include "monitor_104_rhou2_B0_kernel.cu"
#include "monitor_105_rhou2_B0_kernel.cu"
#include "monitor_106_rhou2_B0_kernel.cu"
#include "monitor_107_rhou2_B0_kernel.cu"
#include "monitor_108_rhou2_B0_kernel.cu"
#include "monitor_109_rhou2_B0_kernel.cu"
#include "monitor_110_rhou2_B0_kernel.cu"
#include "opensbliblock00Kernel083_kernel.cu"
