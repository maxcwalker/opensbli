#include "hip/hip_runtime.h"
// Auto-generated at 2024-10-10 09:28:36.252344 by ops-translator legacy

__constant__ int dims_opensbliblock00Kernel016 [3][2];
static int dims_opensbliblock00Kernel016_h [3][2] = {{0}};

//  =============
//  User function
//  =============
__device__ void opensbliblock00Kernel016_gpu(const ACC<double> &rho_B0,
  const ACC<double> &rhou2_B0,
  ACC<double> &u2_B0)
{
   u2_B0(0,0,0) = rhou2_B0(0,0,0)/rho_B0(0,0,0);

}


//  ============================
//  Cuda kernel wrapper function
//  ============================
__global__ void ops_opensbliblock00Kernel016(
double* __restrict arg0,
double* __restrict arg1,
double* __restrict arg2,
int size0,
int size1,
int size2) {
    int idx_z = blockDim.z * blockIdx.z + threadIdx.z;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;

    arg0 += idx_x * 1*1 + idx_y * 1*1 * dims_opensbliblock00Kernel016[0][0] + idx_z * 1*1 * dims_opensbliblock00Kernel016[0][0] * dims_opensbliblock00Kernel016[0][1];
    arg1 += idx_x * 1*1 + idx_y * 1*1 * dims_opensbliblock00Kernel016[1][0] + idx_z * 1*1 * dims_opensbliblock00Kernel016[1][0] * dims_opensbliblock00Kernel016[1][1];
    arg2 += idx_x * 1*1 + idx_y * 1*1 * dims_opensbliblock00Kernel016[2][0] + idx_z * 1*1 * dims_opensbliblock00Kernel016[2][0] * dims_opensbliblock00Kernel016[2][1];

    if (idx_x < size0 && idx_y < size1 && idx_z < size2) {
        const ACC<double> argp0(dims_opensbliblock00Kernel016[0][0], dims_opensbliblock00Kernel016[0][1], arg0);
        const ACC<double> argp1(dims_opensbliblock00Kernel016[1][0], dims_opensbliblock00Kernel016[1][1], arg1);
        ACC<double> argp2(dims_opensbliblock00Kernel016[2][0], dims_opensbliblock00Kernel016[2][1], arg2);
        opensbliblock00Kernel016_gpu(
         argp0, argp1, argp2);
    }

} //End of cuda kernel wrapper function

//  ==================
//  Host stub function
//  ==================
#ifndef OPS_LAZY
void ops_par_loop_opensbliblock00Kernel016(char const *name, ops_block block, int dim, int* range,
 ops_arg arg0, ops_arg arg1, ops_arg arg2) {
#else
void ops_par_loop_opensbliblock00Kernel016_execute(ops_kernel_descriptor *desc) {
    ops_block block = desc->block;
    int dim = desc->dim;
    int *range = desc->range;
    ops_arg arg0 = desc->args[0];
    ops_arg arg1 = desc->args[1];
    ops_arg arg2 = desc->args[2];
#endif

//  ======
//  Timing
//  ======
    double __t1, __t2, __c1, __c2;

    ops_arg args[3];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;

#if CHECKPOINTING && !OPS_LAZY
    if (!ops_checkpointing_before(args, 3, range, 13)) return;
#endif

    if (block->instance->OPS_diags > 1) {
        ops_timing_realloc(block->instance, 13, "opensbliblock00Kernel016");
        block->instance->OPS_kernels[13].count++;
        ops_timers_core(&__c1, &__t1);
    }

//  =================================================
//  compute locally allocated range for the sub-block
//  =================================================
    int start_indx[3];
    int end_indx[3];
#ifdef OPS_MPI
    int arg_idx[3];
#endif

#if defined(OPS_LAZY) || !defined(OPS_MPI)
    for (int n = 0; n < 3; n++) {
        start_indx[n] = range[2*n];
        end_indx[n]   = range[2*n+1];
    }
#else
    if (compute_ranges(args, 3, block, range, start_indx, end_indx, arg_idx) < 0) return;
#endif

    int xdim0 = args[0].dat->size[0];
    int ydim0 = args[0].dat->size[1];
    int xdim1 = args[1].dat->size[0];
    int ydim1 = args[1].dat->size[1];
    int xdim2 = args[2].dat->size[0];
    int ydim2 = args[2].dat->size[1];

    if (xdim0 != dims_opensbliblock00Kernel016_h[0][0] || ydim0 != dims_opensbliblock00Kernel016_h[0][1] || xdim1 != dims_opensbliblock00Kernel016_h[1][0] || ydim1 != dims_opensbliblock00Kernel016_h[1][1] || xdim2 != dims_opensbliblock00Kernel016_h[2][0] || ydim2 != dims_opensbliblock00Kernel016_h[2][1]) {
        dims_opensbliblock00Kernel016_h[0][0] = xdim0;
        dims_opensbliblock00Kernel016_h[0][1] = ydim0;
        dims_opensbliblock00Kernel016_h[1][0] = xdim1;
        dims_opensbliblock00Kernel016_h[1][1] = ydim1;
        dims_opensbliblock00Kernel016_h[2][0] = xdim2;
        dims_opensbliblock00Kernel016_h[2][1] = ydim2;

        cutilSafeCall(block->instance->ostream(), hipMemcpyToSymbol(HIP_SYMBOL( dims_opensbliblock00Kernel016), dims_opensbliblock00Kernel016_h, sizeof(dims_opensbliblock00Kernel016)));
    }


    int x_size = MAX(0,end_indx[0]-start_indx[0]);
    int y_size = MAX(0,end_indx[1]-start_indx[1]);
    int z_size = MAX(0,end_indx[2]-start_indx[2]);

    dim3 grid( (x_size-1)/block->instance->OPS_block_size_x+ 1, (y_size-1)/block->instance->OPS_block_size_y + 1, (z_size-1)/block->instance->OPS_block_size_z +1);
    dim3 tblock(block->instance->OPS_block_size_x,block->instance->OPS_block_size_y,block->instance->OPS_block_size_z);



    long long int dat0 = (block->instance->OPS_soa ? args[0].dat->type_size : args[0].dat->elem_size);
    long long int dat1 = (block->instance->OPS_soa ? args[1].dat->type_size : args[1].dat->elem_size);
    long long int dat2 = (block->instance->OPS_soa ? args[2].dat->type_size : args[2].dat->elem_size);

    char *p_a[3];

//  =======================
//  set up initial pointers
//  =======================
    long long int base0 = args[0].dat->base_offset + 
             dat0 * 1 * (start_indx[0] * args[0].stencil->stride[0]);
    base0 = base0 + dat0 *
                       args[0].dat->size[0] *
                      (start_indx[1] * args[0].stencil->stride[1]);
    base0 = base0 + dat0 *
                       args[0].dat->size[0] *
                       args[0].dat->size[1] *
                      (start_indx[2] * args[0].stencil->stride[2]);
    p_a[0] = (char *)args[0].data_d + base0;

    long long int base1 = args[1].dat->base_offset + 
             dat1 * 1 * (start_indx[0] * args[1].stencil->stride[0]);
    base1 = base1 + dat1 *
                       args[1].dat->size[0] *
                      (start_indx[1] * args[1].stencil->stride[1]);
    base1 = base1 + dat1 *
                       args[1].dat->size[0] *
                       args[1].dat->size[1] *
                      (start_indx[2] * args[1].stencil->stride[2]);
    p_a[1] = (char *)args[1].data_d + base1;

    long long int base2 = args[2].dat->base_offset + 
             dat2 * 1 * (start_indx[0] * args[2].stencil->stride[0]);
    base2 = base2 + dat2 *
                       args[2].dat->size[0] *
                      (start_indx[1] * args[2].stencil->stride[1]);
    base2 = base2 + dat2 *
                       args[2].dat->size[0] *
                       args[2].dat->size[1] *
                      (start_indx[2] * args[2].stencil->stride[2]);
    p_a[2] = (char *)args[2].data_d + base2;

//  =============
//  Halo exchange
//  =============
#ifndef OPS_LAZY
    ops_H_D_exchanges_device(args, 3);
    ops_halo_exchanges(args, 3, range);
#endif

    if (block->instance->OPS_diags > 1) {
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[13].mpi_time += __t2 - __t1;
    }


//call kernel wrapper function, passing in pointers to data
    if (x_size > 0 && y_size > 0 && z_size > 0) {

        ops_opensbliblock00Kernel016<<<grid, tblock >>> ( 
         (double *)p_a[0], (double *)p_a[1],
         (double *)p_a[2],
        x_size, y_size, z_size);

    }

    cutilSafeCall(block->instance->ostream(), hipGetLastError());

    if (block->instance->OPS_diags>1) {
        cutilSafeCall(block->instance->ostream(), hipDeviceSynchronize());
        ops_timers_core(&__c1, &__t1);
        block->instance->OPS_kernels[13].time += __t1 - __t2;
    }

#ifndef OPS_LAZY
    ops_set_dirtybit_device(args, 3);
    ops_set_halo_dirtybit3(&args[2], range);
#endif

    if (block->instance->OPS_diags > 1) {
        //  ====================
        //  Update kernel record
        //  ====================
        ops_timers_core(&__c2, &__t2);
        block->instance->OPS_kernels[13].mpi_time += __t2 - __t1;
        block->instance->OPS_kernels[13].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg0);
        block->instance->OPS_kernels[13].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg1);
        block->instance->OPS_kernels[13].transfer += ops_compute_transfer(dim, start_indx, end_indx, &arg2);
    }
}

#ifdef OPS_LAZY
void ops_par_loop_opensbliblock00Kernel016(char const *name, ops_block block, int dim, int* range,
 ops_arg arg0, ops_arg arg1, ops_arg arg2)
{
    ops_arg args[3];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;

    create_kerneldesc_and_enque("opensbliblock00Kernel016", args, 3, 13, dim, 1, range, block, ops_par_loop_opensbliblock00Kernel016_execute);
}
#endif
